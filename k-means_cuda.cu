#include "hip/hip_runtime.h"
#include <iostream>
#include "INIReader.h"
#include <fstream>
#include <sstream>
#include <chrono>
#include <cmath>
#include <hip/hip_runtime_api.h>

using namespace std;
using namespace chrono;

static const int POINTS_NUMBER = 4000000;
static const int CLUSTER_NUMBER = 4;
static const int ITERATION_NUMBER = 10;
static const int THREADS_PER_BLOCK = 256;

static const string CONFIG_FILE_PATH = "../config_files/config_sets.ini";
static const string DESIRED_CONFIG = to_string(CLUSTER_NUMBER) + "_cluster";

struct DataPoints {
    float* x;
    float* y;
    float* z;
};

void printCentroids(DataPoints& centroids);
string getNumberString(int number);
bool readDatasetFromFile(DataPoints& dataset);
bool initializeCentroids(DataPoints& centroids, const string& configFilePath, const string& desiredConfig);
void freeDataPoints(DataPoints &dataPoints);
void freeDevDataPoints(float *&devPointX, float *&devPointY, float *&devPointZ);
void allocPointDevMemory(float *&hostPointX, float *&hostPointY, float *&hostPointZ, float *&devPointX, float *&devPointY, float *&devPointZ, int size);
__global__ void assignPointToCluster(const float* devPointX, const float* devPointY, const float* devPointZ, float* devCentroidX, float* devCentroidY, float* devCentroidZ, float* devNewCentroidX, float* devNewCentroidY, float* devNewCentroidZ, float* devClustersSize);
void getNewCentroidsAndClustersSize(DataPoints &centroids, float *clustersSize, const float *devClustersSize, const float *devNewCentroidX, const float *devNewCentroidY, const float *devNewCentroidZ);
void updateGlobalMemory(const DataPoints &centroids, float *devCentroidX, float *devCentroidY, float *devCentroidZ, const float *defaultArray, float *devClustersSize, float *devNewCentroidX, float *devNewCentroidY, float *devNewCentroidZ);

int main() {

    DataPoints dataPoints{};
    if(!readDatasetFromFile(dataPoints)) return -1;

    auto startTime = high_resolution_clock::now();
    float *devPointX, *devPointY, *devPointZ;
    allocPointDevMemory(dataPoints.x, dataPoints.y, dataPoints.z, devPointX, devPointY, devPointZ, POINTS_NUMBER);

    DataPoints centroids{};
    if (!initializeCentroids(centroids, CONFIG_FILE_PATH, DESIRED_CONFIG)) return -1;
    printCentroids(centroids);

    float *devCentroidX, *devCentroidY, *devCentroidZ;
    allocPointDevMemory(centroids.x, centroids.y, centroids.z, devCentroidX, devCentroidY, devCentroidZ, CLUSTER_NUMBER);

    auto* defaultArray = new float[CLUSTER_NUMBER]();
    auto* clustersSize = new float[CLUSTER_NUMBER]();
    float* devClustersSize;
    hipMalloc((void**)&devClustersSize, CLUSTER_NUMBER*sizeof(float));
    hipMemcpy(devClustersSize, defaultArray, CLUSTER_NUMBER*sizeof(float), hipMemcpyHostToDevice);

    float *devNewCentroidX, *devNewCentroidY, *devNewCentroidZ;
    allocPointDevMemory(defaultArray, defaultArray, defaultArray, devNewCentroidX, devNewCentroidY, devNewCentroidZ, CLUSTER_NUMBER);

    auto startTime1 = high_resolution_clock::now();

    for (int iteration = 0; iteration < ITERATION_NUMBER; iteration++) {
        cout << endl << "Iteration " << iteration + 1 << ":" << endl;

        dim3 dimBlock(THREADS_PER_BLOCK);
        dim3 dimGrid((POINTS_NUMBER+THREADS_PER_BLOCK-1) / THREADS_PER_BLOCK);
        assignPointToCluster <<<dimGrid,dimBlock,7*CLUSTER_NUMBER*sizeof(float)>>> (devPointX,devPointY,devPointZ,devCentroidX,devCentroidY,devCentroidZ,devNewCentroidX,devNewCentroidY,devNewCentroidZ,devClustersSize);
        hipDeviceSynchronize();

        getNewCentroidsAndClustersSize(centroids, clustersSize, devClustersSize, devNewCentroidX, devNewCentroidY, devNewCentroidZ);

        for (int i = 0; i < CLUSTER_NUMBER; i++) {
            centroids.x[i] = centroids.x[i] / clustersSize[i];
            centroids.y[i] = centroids.y[i] / clustersSize[i];
            centroids.z[i] = centroids.z[i] / clustersSize[i];
        }

        updateGlobalMemory(centroids, devCentroidX, devCentroidY, devCentroidZ, defaultArray, devClustersSize, devNewCentroidX, devNewCentroidY, devNewCentroidZ);

        cout << endl;
        for (int i = 0; i < CLUSTER_NUMBER; i++) {
            cout << "Cluster" << i + 1 << " size: " << static_cast<int>(clustersSize[i]) << endl;
        }

        cout << endl;
        printCentroids(centroids);
    }

    auto endTime = high_resolution_clock::now();
    auto time = duration_cast<microseconds>(endTime - startTime).count() / 1000.f;
    cout << "Duration pre global memory allocation: " << time << " ms" << endl;
    auto time1 = duration_cast<microseconds>(endTime - startTime1).count() / 1000.f;
    cout << "Duration post global memory allocation: " << time1 << " ms" << endl;

    freeDataPoints(dataPoints);
    freeDevDataPoints(devPointX,devPointY,devPointZ);
    freeDataPoints(centroids);
    freeDevDataPoints(devCentroidX,devCentroidY,devCentroidZ);
    delete[] defaultArray;
    delete[] clustersSize;
    freeDevDataPoints(devNewCentroidX,devNewCentroidY,devNewCentroidZ);
    hipFree(devClustersSize);
    devClustersSize = nullptr;

    return 0;
}

void updateGlobalMemory(const DataPoints &centroids, float *devCentroidX, float *devCentroidY, float *devCentroidZ, const float *defaultArray, float *devClustersSize, float *devNewCentroidX, float *devNewCentroidY, float *devNewCentroidZ) {
    hipMemcpy(devCentroidX, centroids.x, CLUSTER_NUMBER * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devCentroidY, centroids.y, CLUSTER_NUMBER * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devCentroidZ, centroids.z, CLUSTER_NUMBER * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devNewCentroidX, defaultArray, CLUSTER_NUMBER*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devNewCentroidY, defaultArray, CLUSTER_NUMBER*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devNewCentroidZ, defaultArray, CLUSTER_NUMBER*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devClustersSize, defaultArray, CLUSTER_NUMBER*sizeof(float), hipMemcpyHostToDevice);
}

void getNewCentroidsAndClustersSize(DataPoints &centroids, float *clustersSize, const float *devClustersSize, const float *devNewCentroidX, const float *devNewCentroidY, const float *devNewCentroidZ) {
    hipMemcpy(centroids.x, devNewCentroidX, CLUSTER_NUMBER * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(centroids.y, devNewCentroidY, CLUSTER_NUMBER * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(centroids.z, devNewCentroidZ, CLUSTER_NUMBER * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(clustersSize,devClustersSize, CLUSTER_NUMBER * sizeof(float), hipMemcpyDeviceToHost);
}

void allocPointDevMemory(float *&hostPointX, float *&hostPointY, float *&hostPointZ, float *&devPointX, float *&devPointY, float *&devPointZ, int size) {
    hipMalloc((void**)&devPointX, size * sizeof(float));
    hipMalloc((void**)&devPointY, size * sizeof(float));
    hipMalloc((void**)&devPointZ, size * sizeof(float));
    hipMemcpy(devPointX, hostPointX, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devPointY, hostPointY, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devPointZ, hostPointZ, size * sizeof(float), hipMemcpyHostToDevice);
}

void freeDataPoints(DataPoints &dataPoints) {
    free(dataPoints.x);
    free(dataPoints.y);
    free(dataPoints.z);
    dataPoints.x = nullptr;
    dataPoints.y = nullptr;
    dataPoints.z = nullptr;
}

void freeDevDataPoints(float *&devPointX, float *&devPointY, float *&devPointZ){
    hipFree(devPointX);
    hipFree(devPointY);
    hipFree(devPointZ);
    devPointX = nullptr;
    devPointY = nullptr;
    devPointZ = nullptr;
}

bool readDatasetFromFile(DataPoints& dataset) {
    dataset.x = (float*)malloc(POINTS_NUMBER* sizeof(float));
    dataset.y = (float*)malloc(POINTS_NUMBER* sizeof(float));
    dataset.z = (float*)malloc(POINTS_NUMBER* sizeof(float));
    string path = "../datasets/generated_blob_dataset_" + getNumberString(POINTS_NUMBER) + ".csv";
    ifstream file(path);
    if (file.is_open()) {
        string line;
        cout << "Reading the dataset..." << endl;
        int i = 0;
        while (getline(file, line)) {
            istringstream coordinates(line);
            float x;
            float y;
            float z;
            char delimiter1;
            char delimiter2;
            if (coordinates >> x >> delimiter1 >> y >> delimiter2 >> z) {
                dataset.x[i] = x;
                dataset.y[i] = y;
                dataset.z[i] = z;
                i++;
            }
        }
        file.close();
        cout << "Dataset loaded from " << path << endl;
        return true;
    } else {
        cerr << "Error: Unable to open file " << path << endl;
        freeDataPoints(dataset);
        return false;
    }
}

string getNumberString(int number) {
    switch (number) {
        case 4000:
            return "4k";
        case 40000:
            return "40k";
        case 400000:
            return "400k";
        case 4000000:
            return "4m";
        case 40000000:
            return "40m";
        default:
            return "";
    }
}

bool initializeCentroids(DataPoints& centroids, const string& configFilePath, const string& desiredConfig) {
    INIReader reader(configFilePath);
    if (reader.ParseError() < 0) {
        cerr << "Error loading config file\n";
        return false;
    }
    centroids.x = (float*)malloc(CLUSTER_NUMBER * sizeof(float));
    centroids.y = (float*)malloc(CLUSTER_NUMBER * sizeof(float));
    centroids.z = (float*)malloc(CLUSTER_NUMBER * sizeof(float));
    for(int i=0; i < CLUSTER_NUMBER; i++)  {
        istringstream coordinates(reader.Get(desiredConfig, "centroid" + to_string(i), ""));
        float x;
        float y;
        float z;
        char delimiter1;
        char delimiter2;
        if (coordinates >> x >> delimiter1 >> y >> delimiter2 >> z){
            centroids.x[i] = x;
            centroids.y[i] = y;
            centroids.z[i] = z;
        }
    }
    return true;
}

void printCentroids(DataPoints& centroids) {
    for (int i=0; i<CLUSTER_NUMBER; i++) {
        cout << "(" << centroids.x[i] << ", " << centroids.y[i] << ", " << centroids.z[i] << ")" << endl;
    }
}

__global__ void assignPointToCluster(const float* devPointX, const float* devPointY, const float* devPointZ, float* devCentroidX, float* devCentroidY, float* devCentroidZ, float* devNewCentroidX, float* devNewCentroidY, float* devNewCentroidZ, float* devClustersSize) {
    extern __shared__ float sharedMem[];
    int tid = blockIdx.x*blockDim.x+threadIdx.x;
    if (threadIdx.x == 0) {
        for (int i = 0; i < CLUSTER_NUMBER; i++) {
            sharedMem[i] = devCentroidX[i];
            sharedMem[CLUSTER_NUMBER + i] = devCentroidY[i];
            sharedMem[2*CLUSTER_NUMBER + i] = devCentroidZ[i];
            sharedMem[3*CLUSTER_NUMBER + i] = 0;
            sharedMem[4*CLUSTER_NUMBER + i] = 0;
            sharedMem[5*CLUSTER_NUMBER + i] = 0;
            sharedMem[6*CLUSTER_NUMBER + i] = 0;
        }
    }
    __syncthreads();
    if (tid < POINTS_NUMBER) {
        float x = devPointX[tid];
        float y = devPointY[tid];
        float z = devPointZ[tid];
        float shortest_distance = sqrt(
                pow(sharedMem[0] - x, 2) + pow(sharedMem[CLUSTER_NUMBER] - y, 2) +
                pow(sharedMem[2*CLUSTER_NUMBER] - z, 2));
        int clusterType = 0;
        for (int i = 1; i < CLUSTER_NUMBER; i++) {
            float centroid_distance = sqrt(
                    pow(sharedMem[i] - x, 2) + pow(sharedMem[CLUSTER_NUMBER + i] - y, 2) +
                    pow(sharedMem[2*CLUSTER_NUMBER + i] - z, 2));
            if (centroid_distance < shortest_distance) {
                shortest_distance = centroid_distance;
                clusterType = i;
            }
        }
        atomicAdd(&(sharedMem[3*CLUSTER_NUMBER + clusterType]), x);
        atomicAdd(&(sharedMem[4*CLUSTER_NUMBER + clusterType]), y);
        atomicAdd(&(sharedMem[5*CLUSTER_NUMBER + clusterType]), z);
        atomicAdd(&(sharedMem[6*CLUSTER_NUMBER + clusterType]), 1);
    }
    __syncthreads();
    if (threadIdx.x==0) {
        for (int i = 0; i < CLUSTER_NUMBER; i++) {
            atomicAdd(&(devNewCentroidX[i]), sharedMem[3*CLUSTER_NUMBER + i]);
            atomicAdd(&(devNewCentroidY[i]), sharedMem[4*CLUSTER_NUMBER + i]);
            atomicAdd(&(devNewCentroidZ[i]), sharedMem[5*CLUSTER_NUMBER + i]);
            atomicAdd(&(devClustersSize[i]), sharedMem[6*CLUSTER_NUMBER + i]);
        }
    }
}